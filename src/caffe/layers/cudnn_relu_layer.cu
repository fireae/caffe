
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
<<<<<<< HEAD
#include <vector>

#include "caffe/layers/cudnn_relu_layer.hpp"
=======
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
>>>>>>> caffe-yolo/master

namespace caffe {

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
<<<<<<< HEAD
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
=======
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        CUDNN_ACTIVATION_RELU,
>>>>>>> caffe-yolo/master
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
<<<<<<< HEAD
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
=======
>>>>>>> caffe-yolo/master
}

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Backward_gpu(top, propagate_down, bottom);
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
<<<<<<< HEAD
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
=======
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        CUDNN_ACTIVATION_RELU,
>>>>>>> caffe-yolo/master
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
<<<<<<< HEAD
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
=======
>>>>>>> caffe-yolo/master
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNReLULayer);

}  // namespace caffe
#endif
