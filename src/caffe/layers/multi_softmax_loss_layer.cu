#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/loss_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MultiSoftmaxLossForwardGPU(const int nthreads,
	  const Dtype* prob_data, const Dtype* label, Dtype* loss,
	  const int num, const int dim, const int spatial_dim,
	  const bool has_ignore_label_, const int ignore_label_, Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_num = dim / spatial_dim;
    for (int c = 0; c < label_num; ++c) {
      int label_index = n * dim + c * spatial_dim + s;
      if (label[label_index] == 1 && c != ignore_label_) {
	loss[label_index] = -log(max(prob_data[label_index], Dtype(FLT_MIN)));
	counts[label_index] = 1;
      } else {
	loss[label_index] = 0;
	counts[label_index] = 0;
      }
    }
  }
}

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Forward_gpu (
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* counts = prob_.mutable_gpu_diff();
  MultiSoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(prob_.count(), loss_data, &loss);
  if (normalize_) {
    Dtype count;
    caffe_gpu_asum(prob_.count(), counts, &count);
    loss /= count;
  } else {
    loss /= outer_num_;
  }
  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void MultiSoftmaxLossBackwardGPU(const int nthreads, const Dtype* label,
	  Dtype* bottom_diff, const int num, const int dim, const int spatial_dim,
	  const bool has_ignore_label_, const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;
  
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    int truth_count = 0;
    for (int c = 0; c < channels; ++c) {
      if (label[n * dim + c * spatial_dim + s] == 1 && c != ignore_label_) {
	truth_count++;
      }
    }

    for (int c = 0; c < channels; ++c) {
      int label_index = n * dim + c * spatial_dim + s;
      bottom_diff[label_index] *= truth_count;
      if (label[label_index] == 1 && c != ignore_label_) {
	bottom_diff[label_index] -= 1;
	counts[label_index] = 1;
      }
    }

  }
}

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Backward_gpu (const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs. ";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    Dtype* counts = prob_.mutable_gpu_diff();
    MultiSoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
	CAFFE_CUDA_NUM_THREADS>>>(nthreads, label, bottom_diff, outer_num_,
	dim, inner_num_, has_ignore_label_, ignore_label_, counts);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(prob_.count(), counts, &count);
      caffe_gpu_scal(prob_.count(), loss_weight / count, bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiSoftmaxWithLossLayer);

}